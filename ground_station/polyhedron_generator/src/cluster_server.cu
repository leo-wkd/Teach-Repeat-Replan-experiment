#include "hip/hip_runtime.h"
#include "polyhedron_generator/cluster_server.cuh"
#include "polyhedron_generator/cluster_engine.cuh"
#include "polyhedron_generator/cluster_engine_cpu.h"

using namespace std;

#define DEBUG_VIS 0
#define DEBUG_INFO_VERBOSE_LEVEL_1 0
#define DEBUG_INFO_VERBOSE_LEVEL_2 0

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

void cudaPolytopeGeneration::mapUpload()
{   
    hipMemcpy(d_map_data, map_data, sizeof(uint8_t) * _grid_num, hipMemcpyHostToDevice);  
}

void cudaPolytopeGeneration::insideFlagUpload()
{   
    hipMemcpy(d_inside_data, inside_data,  sizeof(uint8_t) * _grid_num, hipMemcpyHostToDevice);  
}

void cudaPolytopeGeneration::mapClear()
{   
    memset(map_data,     (uint8_t)0, _grid_num*sizeof(uint8_t)); 
}

void cudaPolytopeGeneration::flagClear()
{   
// #####
    memset(use_data,     (uint8_t)0, _grid_num*sizeof(uint8_t)); 
    memset(invalid_data, (uint8_t)0, _grid_num*sizeof(uint8_t)); 
    memset(inside_data,  (uint8_t)0, _grid_num*sizeof(uint8_t)); 
}

void cudaPolytopeGeneration::setObs(const int & id_x, const int & id_y, const int & id_z)
{
    map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] = (uint8_t)1;
}

void cudaPolytopeGeneration::setFr(const int & id_x, const int & id_y, const int & id_z)
{
    map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] = (uint8_t)0;
}

void cudaPolytopeGeneration::setObs(const int & idx)
{
    map_data[ idx ] = (uint8_t)1;
}

void cudaPolytopeGeneration::setFr(const int & idx)
{
    map_data[ idx ] = (uint8_t)0;
}

void cudaPolytopeGeneration::setVertexInitIndex(int * vertex_idx, int min_x, int min_y, int min_z, int max_x, int max_y, int max_z)
{
    vertex_idx[0]    = vertex_idx[1]    = vertex_idx[4]    = vertex_idx[5]    = max_x;
    vertex_idx[2]    = vertex_idx[3]    = vertex_idx[6]    = vertex_idx[7]    = min_x;

    vertex_idx[1+8]  = vertex_idx[2+8]  = vertex_idx[5+8]  = vertex_idx[6+8]  = max_y;
    vertex_idx[0+8]  = vertex_idx[3+8]  = vertex_idx[4+8]  = vertex_idx[7+8]  = min_y;

    vertex_idx[0+16] = vertex_idx[1+16] = vertex_idx[2+16] = vertex_idx[3+16] = max_x;
    vertex_idx[4+16] = vertex_idx[5+16] = vertex_idx[6+16] = vertex_idx[7+16] = min_z;
}

void cudaPolytopeGeneration::getVoxelsInCube( 
    int * vertex_idx, vector<int> & cube_grid_x, vector<int> & cube_grid_y, vector<int> & cube_grid_z, 
    uint8_t * inside_data, int map_yz_size, int map_z_size )
{
    int id_x, id_y, id_z;
    for(id_x = vertex_idx[0 + 7]; id_x <= vertex_idx[0 + 1]; id_x++ )
    {   
        for(id_y = vertex_idx[8 + 7]; id_y <= vertex_idx[8 + 1]; id_y++ )
        {
            for(id_z = vertex_idx[16 + 7]; id_z <= vertex_idx[16 + 1]; id_z++ )
            {   
                cube_grid_x.push_back(id_x);
                cube_grid_y.push_back(id_y);
                cube_grid_z.push_back(id_z);

                inside_data[id_x * map_yz_size + id_y * map_z_size + id_z] = (uint8_t)1;
            }
        }
    }
}

void cudaPolytopeGeneration::paramSet( bool is_gpu_on_stage_1, bool is_gpu_on_stage_2, bool is_cluster_on,
    const int & max_x_id,  const int & max_y_id,  const int & max_z_id, double resolution, double itr_inflate_max_, double itr_cluster_max_ )
{   
    _is_gpu_on_stage_1 = is_gpu_on_stage_1;
    _is_gpu_on_stage_2 = is_gpu_on_stage_2;

    _is_cluster_on     = is_cluster_on;

    _resolution = resolution;
    _max_x_id  = max_x_id;
    _max_y_id  = max_y_id;
    _max_z_id  = max_z_id;
    _max_yz_id = max_y_id * max_z_id;
    _grid_num = max_x_id * max_y_id * max_z_id;

    if(is_cluster_on){
        itr_inflate_max = itr_inflate_max_;
        itr_cluster_max = itr_cluster_max_;
    }
    else{
        itr_inflate_max = 1000;
        itr_cluster_max = 0;   
    }

    _cluster_buffer_size   = 50000;
    _candidate_buffer_size = 10000;
    _cluster_buffer_size_square = _candidate_buffer_size * _candidate_buffer_size;

//###### Host data
    map_data     = new uint8_t[_grid_num];
    use_data     = new uint8_t[_grid_num];
    invalid_data = new uint8_t[_grid_num];
    inside_data  = new uint8_t[_grid_num];

    candidate_result  = new bool[_candidate_buffer_size];
    active_xyz_id     = new int[_candidate_buffer_size * 3];
    vertex_idx        = new int[24];
    vertex_idx_lst    = new int[24];

// ######
  //### host data 
    checkCuda(hipHostMalloc((void**)&h_inflate_result, sizeof(bool) * 1                          ));
    checkCuda(hipHostMalloc((void**)&cluster_xyz_id,   sizeof(int)  * _cluster_buffer_size   * 3 ));
    checkCuda(hipHostMalloc((void**)&candidate_xyz_id, sizeof(int)  * _candidate_buffer_size * 3 ));
    checkCuda(hipHostMalloc((void**)&h_can_can_result, sizeof(bool) * _cluster_buffer_size_square));
    checkCuda(hipHostMalloc((void**)&h_can_clu_result, sizeof(bool) * _candidate_buffer_size     ));

   //### Device data    
    hipMalloc((void**)&d_map_data,         sizeof(uint8_t)  * _grid_num);
    hipMalloc((void**)&d_inside_data,      sizeof(uint8_t)  * _grid_num);
    hipMalloc((void**)&d_vertex_idx,       sizeof(int)  * 24);
    hipMalloc((void**)&d_result,           sizeof(bool) * _candidate_buffer_size * (_candidate_buffer_size + _cluster_buffer_size));     

    hipMalloc((void**)&d_inflate_result,   sizeof(bool));
    hipMalloc((void**)&d_cluster_xyz_id,   sizeof(int)  * _cluster_buffer_size   * 3);
    hipMalloc((void**)&d_candidate_xyz_id, sizeof(int)  * _candidate_buffer_size * 3);
    hipMalloc((void**)&d_can_can_result,   sizeof(bool) * _cluster_buffer_size_square );
    hipMalloc((void**)&d_can_clu_result,   sizeof(bool) * _candidate_buffer_size );

    blocks_cube.x  = 128;
    threads_cube.x = 128;

    inf_step = 1;
    mapClear();
}

void cudaPolytopeGeneration::inflateX_n(int * vertex_idx)
{
    // X- now is the back side : (p4 -- p3 -- p7 -- p8) face
    if( vertex_idx[0 + 3] == 0 ) return;     

    int id_x, id_y, id_z;
    id_x = vertex_idx[0 + 3] - 1;   
    for(id_y = vertex_idx[8 + 3]; id_y <= vertex_idx[8 + 2]; id_y++ )
    {
        for(id_z = vertex_idx[16 + 7]; id_z <= vertex_idx[16 + 3]; id_z++ )
        {
            if( map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] > 0 )    
            {   
                return;
            }
        }
    }

    vertex_idx[2] -= inf_step;  vertex_idx[3] -= inf_step; vertex_idx[6] -= inf_step; vertex_idx[7] -= inf_step;
}

void cudaPolytopeGeneration::inflateX_p(int * vertex_idx)
{
    // X + now is the front side : (p1 -- p2 -- p6 -- p5) face
    if( vertex_idx[0 + 0] == _max_x_id - 1 ) return;     

    int id_x, id_y, id_z;
    id_x = vertex_idx[0 + 0] + 1;   
    for(id_y = vertex_idx[8 + 0]; id_y <= vertex_idx[8 + 1]; id_y++ )
    {
        for(id_z = vertex_idx[16 + 4]; id_z <= vertex_idx[16 + 0]; id_z++ )
        {
            if( map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] > 0 )    
            {   
                return;
            }
        }
    }

    vertex_idx[0] += inf_step;  vertex_idx[1]  += inf_step; vertex_idx[4]  += inf_step; vertex_idx[5]  += inf_step; 
}

void cudaPolytopeGeneration::inflateY_n(int * vertex_idx)
{
    // Y- now is the left side : (p1 -- p4 -- p8 -- p5) face sweep
    if( vertex_idx[8 + 0] == 0 ) return;     

    int id_x, id_y, id_z;
    id_y = vertex_idx[8 + 0] - 1;   
    for(id_x = vertex_idx[0 + 3]; id_x <= vertex_idx[0 + 0]; id_x++ )
    {
        for(id_z = vertex_idx[16 + 4]; id_z <= vertex_idx[16 + 0]; id_z++ )
        {
            if( map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] > 0 )    
            {   
                return;
            }
        }
    }

    vertex_idx[8] -= inf_step;  vertex_idx[11] -= inf_step; vertex_idx[12] -= inf_step; vertex_idx[15] -= inf_step;
}

void cudaPolytopeGeneration::inflateY_p(int * vertex_idx)
{
    // Y+ now is the right side : (p2 -- p3 -- p7 -- p6) face
    if( vertex_idx[8 + 1] == _max_y_id - 1 ) return;     

    int id_x, id_y, id_z;
    id_y = vertex_idx[8 + 1] + 1;   
    for(id_x = vertex_idx[0 + 2]; id_x <= vertex_idx[0 + 1]; id_x++ )
    {
        for(id_z = vertex_idx[16 + 5]; id_z <= vertex_idx[16 + 1]; id_z++ )
        {
            if( map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] > 0 )    
            {   
                return;
            }
        }
    }

    vertex_idx[9] += inf_step;  vertex_idx[10] += inf_step; vertex_idx[13] += inf_step; vertex_idx[14] += inf_step;
}

void cudaPolytopeGeneration::inflateZ_n(int * vertex_idx)
{   
    // Z- now is the below side : (p5 -- p6 -- p7 -- p8) face
    if( vertex_idx[16 + 4] == 0 ) return;     

    int id_x, id_y, id_z;
    id_z = vertex_idx[16 + 4] - 1; 

    for(id_x = vertex_idx[0 + 7]; id_x <= vertex_idx[0 + 4]; id_x++ )
    {
        for(id_y = vertex_idx[8 + 4]; id_y <= vertex_idx[8 + 5]; id_y++ )
        {
            if( map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] > 0 )    
            {   
                return;
            }
        }
    }

    vertex_idx[20] -= inf_step; vertex_idx[21] -= inf_step; vertex_idx[22] -= inf_step; vertex_idx[23] -= inf_step;
}

void cudaPolytopeGeneration::inflateZ_p(int * vertex_idx)
{ 
    // Z+ now is the above side : (p1 -- p2 -- p3 -- p4) face
    if( vertex_idx[16 + 0] == _max_z_id - 1 ) return;     

    int id_x, id_y, id_z;
    id_z = vertex_idx[16 + 0] + 1; 

    for(id_x = vertex_idx[0 + 3]; id_x <= vertex_idx[0 + 0]; id_x++ )
    {
        for(id_y = vertex_idx[8 + 0]; id_y <= vertex_idx[8 + 1]; id_y++ )
        {
            if( map_data[ id_x * _max_yz_id + id_y * _max_z_id + id_z ] > 0 )    
            {   
                return;
            }
        }
    }

    vertex_idx[16] += inf_step; vertex_idx[17] += inf_step; vertex_idx[18] += inf_step; vertex_idx[19] += inf_step; 
}

void cudaPolytopeGeneration::cubeInflation_cpu( int * vertex_idx_lst, int * vertex_idx )
{   
    int iter = 0;

    while( iter < itr_inflate_max )
    {   
        inflateY_n(vertex_idx);
        inflateY_p(vertex_idx);
        inflateX_n(vertex_idx);
        inflateX_p(vertex_idx);
        inflateZ_n(vertex_idx);
        inflateZ_p(vertex_idx);

        bool is_inflate_conti = false;
        for(int vtx = 0; vtx < 8; vtx ++)
        {
            if( (vertex_idx_lst[vtx] != vertex_idx[vtx]) || (vertex_idx_lst[vtx + 8] != vertex_idx[vtx + 8]) || (vertex_idx_lst[vtx + 16] != vertex_idx[vtx + 16]) )
            {
                is_inflate_conti = true;
                break;
            }
        }

        if(is_inflate_conti == false)
            break;

        for(int vtx = 0; vtx < 8; vtx ++)
        { 
            vertex_idx_lst[vtx +  0] = vertex_idx[vtx +  0];
            vertex_idx_lst[vtx +  8] = vertex_idx[vtx +  8];
            vertex_idx_lst[vtx + 16] = vertex_idx[vtx + 16];
        }

        iter ++;
    }

}

void cudaPolytopeGeneration::cubeInflation_gpu( int * vertex_idx_lst, int * vertex_idx, 
                                                double & time_upload_cube,  double & time_download_cube, double & time_cuda_cube )
{
    int iter = 0;
    while( iter < itr_inflate_max )
    {   
        //ROS_WARN(" Cube Geneartion;  Main Iteration, iter: %d", iter);
        // in 6 directions, inflate one step sequtially
        for(int dir = 0; dir < 6; dir++)
        {   
            bool is_dir_max = false;
            switch(dir) 
            { 
                case 0: //direction Y -
                    if( vertex_idx[8] == 0)              is_dir_max = true;
                    break;
                case 1: //direction Y +
                    if( vertex_idx[9] == _max_y_id - 1)  is_dir_max = true;
                    break; 
                case 2: //direction X -
                    if( vertex_idx[3] == 0)              is_dir_max = true;
                    break; 
                case 3: //direction X +
                    if( vertex_idx[0] == _max_x_id - 1)  is_dir_max = true;
                    break; 
                case 4: //direction Z -
                    if( vertex_idx[20] == 0)             is_dir_max = true;
                    break; 
                case 5: //direction Z +  
                    if( vertex_idx[16] == _max_z_id - 1) is_dir_max = true;
                    break;  
                default: 
                    break; 
            }

            if( is_dir_max == true ) 
                continue;

            // update the vertex index in GPU's memory
            ros::Time time_1_cube_vertex_upload = ros::Time::now();
            hipMemcpy(d_vertex_idx, vertex_idx, sizeof(int) * 24, hipMemcpyHostToDevice);   
            ros::Time time_2_cube_vertex_upload = ros::Time::now();
            time_upload_cube += (time_2_cube_vertex_upload - time_1_cube_vertex_upload).toSec();

            paraCubeInflation <<< blocks_cube, threads_cube >>> 
            (
                dir, inf_step, d_map_data, _max_yz_id, _max_z_id, 
                d_vertex_idx, d_inflate_result
            );
            hipDeviceSynchronize();
            ros::Time time_2_cube_kernel = ros::Time::now();
            time_cuda_cube += (time_2_cube_kernel - time_2_cube_vertex_upload).toSec();

            // down the result about one direction's inflation
            ros::Time time_1_cube_vertex_download = ros::Time::now();
            hipMemcpy(h_inflate_result, d_inflate_result, sizeof(bool), hipMemcpyDeviceToHost);
            ros::Time time_2_cube_vertex_download = ros::Time::now();
            time_download_cube += (time_2_cube_vertex_download - time_1_cube_vertex_download).toSec();

            // judge whether we should take one step further in this direction
            if( *h_inflate_result == true ) // inflate one step success;
            {   
                switch(dir) 
                { 
                    case 0: //direction Y -
                        vertex_idx[8] -= inf_step;  vertex_idx[11] -= inf_step; vertex_idx[12] -= inf_step; vertex_idx[15] -= inf_step;
                        break;
                    case 1: //direction Y +
                        vertex_idx[9] += inf_step;  vertex_idx[10] += inf_step; vertex_idx[13] += inf_step; vertex_idx[14] += inf_step;
                        break; 
                    case 2: //direction X -
                        vertex_idx[2] -= inf_step;  vertex_idx[3]  -= inf_step; vertex_idx[6]  -= inf_step; vertex_idx[7]  -= inf_step;
                        break; 
                    case 3: //direction X +
                        vertex_idx[0] += inf_step;  vertex_idx[1]  += inf_step; vertex_idx[4]  += inf_step; vertex_idx[5]  += inf_step; 
                        break; 
                    case 4: //direction Z -
                        vertex_idx[20] -= inf_step; vertex_idx[21] -= inf_step; vertex_idx[22] -= inf_step; vertex_idx[23] -= inf_step;
                        break; 
                    case 5: //direction Z +  
                        vertex_idx[16] += inf_step; vertex_idx[17] += inf_step; vertex_idx[18] += inf_step; vertex_idx[19] += inf_step; 
                        break;  
                    default: 
                        break; 
                }
            }
        }

        bool is_inflate_conti = false;
        for(int vtx = 0; vtx < 8; vtx ++)
        {
            if( (vertex_idx_lst[vtx] != vertex_idx[vtx]) || (vertex_idx_lst[vtx + 8] != vertex_idx[vtx + 8]) || (vertex_idx_lst[vtx + 16] != vertex_idx[vtx + 16]) )
            {
                is_inflate_conti = true;
                break;
            }
        }

        if(is_inflate_conti == false)
            break;

        for(int vtx = 0; vtx < 8; vtx ++)
        { 
            vertex_idx_lst[vtx +  0] = vertex_idx[vtx +  0];
            vertex_idx_lst[vtx +  8] = vertex_idx[vtx +  8];
            vertex_idx_lst[vtx + 16] = vertex_idx[vtx + 16];
        }

        iter ++;
    }
}

void cudaPolytopeGeneration::polytopeCluster_cpu( int & cluster_grid_num, int & active_grid_num )
{   
    int candidate_grid_num;
    int can_x_idx, can_y_idx, can_z_idx;

    int itr_cluster_cnt = 0;
    while( itr_cluster_cnt < itr_cluster_max )
    {   
        candidate_grid_num = 0;
        for(int i = 0; i < active_grid_num; i++)
        {
            int cur_idx_x, cur_idx_y, cur_idx_z; 
            int nei_idx_x, nei_idx_y, nei_idx_z;

            cur_idx_x = active_xyz_id[3 * i];
            cur_idx_y = active_xyz_id[3 * i + 1];
            cur_idx_z = active_xyz_id[3 * i + 2];

            use_data[cur_idx_x * _max_yz_id + cur_idx_y * _max_z_id + cur_idx_z] = (uint8_t)1;

            // get all nearby voxels   
            for(int dx = -1; dx < 2; dx++)
            { 
                for(int dy = -1; dy < 2; dy++)
                {   
                    for(int dz = -1; dz < 2; dz++)
                    {   
                        if( dx == 0 && dy == 0 && dz == 0) continue;
                        //if( abs(dx) + abs(dy) + abs(dz) == 1 ) 
                        else
                        { 
                            nei_idx_x = cur_idx_x + dx;
                            nei_idx_y = cur_idx_y + dy;
                            nei_idx_z = cur_idx_z + dz;

                            if(nei_idx_x < 0 || nei_idx_x > _max_x_id - 1 
                            || nei_idx_y < 0 || nei_idx_y > _max_y_id - 1
                            || nei_idx_z < 0 || nei_idx_z > _max_z_id - 1 )
                                continue;

                            if(map_data    [nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 
                            || use_data    [nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 
                            || invalid_data[nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 
                            || inside_data [nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 )
                            {
                                continue;
                            }
                            else
                            {   
                                candidate_xyz_id[3 * candidate_grid_num    ] = nei_idx_x;
                                candidate_xyz_id[3 * candidate_grid_num + 1] = nei_idx_y;
                                candidate_xyz_id[3 * candidate_grid_num + 2] = nei_idx_z;

                                candidate_grid_num ++;
                                use_data[nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] = (uint8_t)1;
                            }
                        }
                    }
                }
            }
        }

        if( candidate_grid_num == 0 ) 
            break;

        active_grid_num = 0;
        for(int i = 0; i < candidate_grid_num; i++ )
        {
            // for each voxel in the candidate set, test if it preserves the convex hull property
            can_x_idx = candidate_xyz_id[3 * i];
            can_y_idx = candidate_xyz_id[3 * i + 1];
            can_z_idx = candidate_xyz_id[3 * i + 2];

            if( serialConvexTest(can_x_idx, can_y_idx, can_z_idx, cluster_grid_num, _max_yz_id, _max_z_id, cluster_xyz_id, inside_data, map_data ) )
            {   
                cluster_xyz_id[3 * cluster_grid_num]     = can_x_idx;
                cluster_xyz_id[3 * cluster_grid_num + 1] = can_y_idx;
                cluster_xyz_id[3 * cluster_grid_num + 2] = can_z_idx;

                active_xyz_id[3 * active_grid_num]       = can_x_idx;
                active_xyz_id[3 * active_grid_num + 1]   = can_y_idx;
                active_xyz_id[3 * active_grid_num + 2]   = can_z_idx;

                cluster_grid_num ++;
                active_grid_num ++;

                inside_data [can_x_idx * _max_yz_id + can_y_idx * _max_z_id + can_z_idx] = 0;
            }
            else
            {   
                invalid_data[can_x_idx * _max_yz_id + can_y_idx * _max_z_id + can_z_idx] = (uint8_t)1;
            }
        }
        
        if( active_grid_num == 0 ) 
            break;

        itr_cluster_cnt ++; 
    }
}

void cudaPolytopeGeneration::polytopeCluster_gpu(int & cluster_grid_num, int active_grid_num, 
                                                 double & time_upload, double & time_download, double & time_cuda)
{   
    int candidate_grid_num;
    ros::Time time_upload_11, time_upload_12, time_upload_21, time_upload_22;
    ros::Time time_download_1, time_download_2;
    ros::Time time_check_candidate_1, time_check_candidate_2;
    ros::Time time_start_gpu_cluster = ros::Time::now();

    double time_check_candidate = 0.0;

    int itr_cluster_cnt = 0;
    while( itr_cluster_cnt < itr_cluster_max )
    {   
        ros::Time time_ite = ros::Time::now();

        candidate_grid_num = 0;
        for(int i = 0; i < active_grid_num; i++)
        {
            int cur_idx_x, cur_idx_y, cur_idx_z; 
            int nei_idx_x, nei_idx_y, nei_idx_z;

            cur_idx_x = active_xyz_id[3 * i];
            cur_idx_y = active_xyz_id[3 * i + 1];
            cur_idx_z = active_xyz_id[3 * i + 2];

            use_data[cur_idx_x * _max_yz_id + cur_idx_y * _max_z_id + cur_idx_z] = (uint8_t)1;

            // get all nearby voxels   
            for(int dx = -1; dx < 2; dx++)
            { 
                for(int dy = -1; dy < 2; dy++)
                {   
                    for(int dz = -1; dz < 2; dz++)
                    {   
                        if( dx == 0 && dy == 0 && dz == 0) continue;
                        //if( abs(dx) + abs(dy) + abs(dz) == 1 ) 
                        else
                        { 
                            nei_idx_x = cur_idx_x + dx;
                            nei_idx_y = cur_idx_y + dy;
                            nei_idx_z = cur_idx_z + dz;

                            if(nei_idx_x < 0 || nei_idx_x > _max_x_id - 1 
                            || nei_idx_y < 0 || nei_idx_y > _max_y_id - 1
                            || nei_idx_z < 0 || nei_idx_z > _max_z_id - 1 )
                                continue;

                            if(map_data    [nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 
                            || use_data    [nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 
                            || invalid_data[nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 
                            || inside_data [nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] == 1 )
                            {
                                continue;
                            }
                            else
                            {   
                                candidate_xyz_id[3 * candidate_grid_num    ] = nei_idx_x;
                                candidate_xyz_id[3 * candidate_grid_num + 1] = nei_idx_y;
                                candidate_xyz_id[3 * candidate_grid_num + 2] = nei_idx_z;

                                candidate_grid_num ++;
                                use_data[nei_idx_x * _max_yz_id + nei_idx_y * _max_z_id + nei_idx_z] = (uint8_t)1;
                            }
                        }
                    }
                }
            }
        }
        
        // upload all candidate grids into the GPU
        time_upload_11 = ros::Time::now();
        hipMemcpy(d_candidate_xyz_id, candidate_xyz_id, 3 * candidate_grid_num * sizeof(int),  hipMemcpyHostToDevice);   

        time_upload_12 = ros::Time::now();
        time_upload += (time_upload_12 - time_upload_11).toSec();

#if DEBUG_VIS
        if(itr_cluster_cnt == 0)
            for(int i = 0; i < candidate_grid_num; i++)
            {
                vis_grid_id_x.push_back(candidate_xyz_id[3 * i + 0]);
                vis_grid_id_y.push_back(candidate_xyz_id[3 * i + 1]);
                vis_grid_id_z.push_back(candidate_xyz_id[3 * i + 2]);
            }
#endif

#if DEBUG_INFO_VERBOSE_LEVEL_1
        cout<<"convex cluster's total     grid num: "<<cluster_grid_num<<endl; 
        cout<<"last iteration's active    grid num: "<<active_grid_num<<endl;
        cout<<"this iteration's candidate grid num: "<<candidate_grid_num<<endl;
#endif

        if( candidate_grid_num == 0 ) break;

        ros::Time time_cuda_1 = ros::Time::now();

        int para_comp_num = candidate_grid_num * (cluster_grid_num + candidate_grid_num);
        threads_cvx.x  = min (1024, candidate_grid_num);
        blocks_cvx.x   = ceil(para_comp_num / threads_cvx.x) + 1;

        paraConvexTest <<< blocks_cvx, threads_cvx >>> 
        (
            d_map_data, d_inside_data, 
            d_candidate_xyz_id, d_cluster_xyz_id, d_result,
            _max_yz_id, _max_z_id,
            candidate_grid_num, cluster_grid_num
        );
        hipDeviceSynchronize();
        
        threads_res_chk.x = min (1024, candidate_grid_num);
        blocks_res_chk.x  = ceil(candidate_grid_num / threads_res_chk.x) + 1;
        //cout<<"blocks_res_chk.x: "<<blocks_res_chk.x<<endl;

        paraResultCheck <<< blocks_res_chk, threads_res_chk >>> (
            d_result, d_can_can_result, d_can_clu_result,
            candidate_grid_num, cluster_grid_num );

        ros::Time time_cuda_2 = ros::Time::now();

// ###   debug data transfering blocking
        int can_can_total_num = candidate_grid_num * (candidate_grid_num - 1) / 2;
        time_download_1 = ros::Time::now();  

        hipMemcpy(h_can_clu_result, d_can_clu_result, candidate_grid_num * sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(h_can_can_result, d_can_can_result, can_can_total_num  * sizeof(bool), hipMemcpyDeviceToHost);
        
        //hipDeviceSynchronize();
        time_download_2 = ros::Time::now();
        time_download += (time_download_2 - time_download_1).toSec();

        time_check_candidate_1 = ros::Time::now();
        int can_x_idx, can_y_idx, can_z_idx;
        //memset(candidate_result, false, candidate_grid_num * sizeof(bool)); 
        std::fill(candidate_result, candidate_result + candidate_grid_num, false);
        
        active_grid_num = 0;
        int cluster_num_lst = cluster_grid_num;
        for(int i = 0; i < candidate_grid_num; i++)
        {   
            bool is_convex = true;
            if(h_can_clu_result[i] == false) 
            {
                is_convex = false;
            }
            else // check its relationship with its prior candidate grids
            {   
                int n = i + 1;
                int can_can_cnt_bias = n * (n - 1) / 2;
                
                for(int j = 0; j < i; j++)
                {
                    if( h_can_can_result[can_can_cnt_bias + j] == false && candidate_result[j] == true )
                    {
                        is_convex = false;
                        break;
                    }
                }
            }

            can_x_idx = candidate_xyz_id[3 * i];
            can_y_idx = candidate_xyz_id[3 * i + 1];
            can_z_idx = candidate_xyz_id[3 * i + 2];

            if(is_convex == true)
            {   
                candidate_result[i] = true;

                cluster_xyz_id[3 * cluster_grid_num]     = can_x_idx;
                cluster_xyz_id[3 * cluster_grid_num + 1] = can_y_idx;
                cluster_xyz_id[3 * cluster_grid_num + 2] = can_z_idx;

                active_xyz_id[3 * active_grid_num]       = can_x_idx;
                active_xyz_id[3 * active_grid_num + 1]   = can_y_idx;
                active_xyz_id[3 * active_grid_num + 2]   = can_z_idx;

                cluster_grid_num ++;
                active_grid_num ++;
            }
            else
            {   
                invalid_data[can_x_idx * _max_yz_id + can_y_idx * _max_z_id + can_z_idx] = (uint8_t)1;
            }
        }
        
        if( active_grid_num == 0 ) break;
        
        time_check_candidate_2 = ros::Time::now();
        time_check_candidate += (time_check_candidate_2 - time_check_candidate_1).toSec();

        time_upload_21 = ros::Time::now();
        //hipMemcpyAsync(&d_cluster_xyz_id[3 * cluster_num_lst], &cluster_xyz_id[3 * cluster_num_lst], 3 * active_grid_num * sizeof(int), hipMemcpyHostToDevice);   
        hipMemcpy(&d_cluster_xyz_id[3 * cluster_num_lst], &cluster_xyz_id[3 * cluster_num_lst], 3 * active_grid_num * sizeof(int), hipMemcpyHostToDevice);   
        time_upload_22 = ros::Time::now();

        time_upload += (time_upload_22 - time_upload_21).toSec();
        time_cuda   += (time_cuda_2 - time_cuda_1).toSec();
        ros::Time time_ite_2 = ros::Time::now();

#if DEBUG_INFO_VERBOSE_LEVEL_2
        ROS_WARN("time totally in one iteration: %f", (time_ite_2 - time_ite).toSec());
        ROS_WARN("time cost in cuda kernel: %f", (time_cuda_2 - time_cuda_1).toSec());
        ROS_WARN("time cost in check resuls: %f", (time_check_candidate_2 - time_check_candidate_1).toSec());
        ROS_WARN("time in data upload_step1: %f", (time_upload_12 - time_upload_11).toSec() );
        ROS_WARN("time in data upload_step2: %f", (time_upload_22 - time_upload_21).toSec() );
        ROS_WARN("time in data download : %f", (time_download_2 - time_download_1).toSec() );
        ROS_WARN("time so far: %f", (time_ite_2 - time_start_gpu_cluster).toSec());
#endif
        itr_cluster_cnt ++; 
    }

}

void cudaPolytopeGeneration::polygonGeneration( 
    vector<int> & cluster_x_idx, vector<int> & cluster_y_idx, vector<int> & cluster_z_idx )
{   
    //cout<<"call polygonGeneration"<<endl;
    flagClear();
    insideFlagUpload();
// ### 
    // Data uploading
    ros::Time time_start_gpu_computing = ros::Time::now();

    ros::Time time_1_cube = ros::Time::now();
    ros::Time time_2_cube = ros::Time::now();
    ros::Time time_1_poly = ros::Time::now();
    ros::Time time_2_poly = ros::Time::now();

    double time_upload_cube   = 0.0;
    double time_download_cube = 0.0;
    double time_cuda_cube = 0.0;

// ###
    // GPU-based cube inflation
    if(cluster_x_idx.size() == 1)
    {
        for(int vtx = 0; vtx < 8; vtx++)
        {   
            vertex_idx[vtx]      = cluster_x_idx[0];
            vertex_idx[vtx + 8 ] = cluster_y_idx[0];
            vertex_idx[vtx + 16] = cluster_z_idx[0];
        }        
    }
    else
    {
        int min_x, max_x, min_y, max_y, min_z, max_z;
        min_x = min_y = min_z = -100000;
        max_x = max_y = max_z = +100000;
        
        for(int i = 0; i < (int)cluster_x_idx.size(); i++)
        {
            min_x = cluster_x_idx[i] < min_x ? cluster_x_idx[i] : min_x;
            min_y = cluster_y_idx[i] < min_y ? cluster_y_idx[i] : min_y;
            min_z = cluster_z_idx[i] < min_z ? cluster_z_idx[i] : min_z;

            max_x = cluster_x_idx[i] > max_x ? cluster_x_idx[i] : max_x;
            max_y = cluster_y_idx[i] > max_y ? cluster_y_idx[i] : max_y;
            max_z = cluster_z_idx[i] > max_z ? cluster_z_idx[i] : max_z;
        }

        setVertexInitIndex(vertex_idx, min_x, min_y, min_z, max_x, max_y, max_z);
    }

    memcpy(vertex_idx_lst, vertex_idx, sizeof(int) * 24);

    time_1_cube = ros::Time::now();
#if _is_gpu_on_stage_1
    cubeInflation_gpu(vertex_idx_lst, vertex_idx, time_upload_cube, time_download_cube, time_cuda_cube);
#else
    cubeInflation_cpu(vertex_idx_lst, vertex_idx);
#endif

    time_2_cube = ros::Time::now();
    cluster_x_idx.clear(); cluster_y_idx.clear(); cluster_z_idx.clear();

    cube_grid_x.clear();
    cube_grid_y.clear();
    cube_grid_z.clear();
    getVoxelsInCube(vertex_idx, cube_grid_x, cube_grid_y, cube_grid_z, inside_data, _max_yz_id, _max_z_id);

    vector<int> cube_outside_grid_x, cube_outside_grid_y, cube_outside_grid_z;
    int tmp_x, tmp_y, tmp_z;

    if(cube_grid_x.size() == 1)
    {
        cube_outside_grid_x.push_back(cube_grid_x[0]);     
        cube_outside_grid_y.push_back(cube_grid_y[0]);     
        cube_outside_grid_z.push_back(cube_grid_z[0]);    

    }
    else
    {
        for(int i = 0; i < (int)cube_grid_x.size(); i++ )
        {   
            int idx = cube_grid_x[i] * _max_yz_id + cube_grid_y[i] * _max_z_id + cube_grid_z[i];
            
            use_data[idx] = (uint8_t)1;

            int is_inside = 1;       
            for(int dx = -1; dx < 2; dx++)
            {   
                for(int dy = -1; dy < 2; dy++)
                {   
                    for(int dz = -1; dz < 2; dz++)
                    {   
                        if(dx == 0 && dy == 0 && dz == 0)
                            continue;

                        tmp_x = cube_grid_x[i] + dx;
                        tmp_y = cube_grid_y[i] + dy;
                        tmp_z = cube_grid_z[i] + dz;

                        if( tmp_x >= 0 && tmp_x < _max_x_id && tmp_y >= 0 && tmp_y < _max_y_id && tmp_z >= 0 && tmp_z < _max_z_id )
                        {
                            int idx_tmp = tmp_x * _max_yz_id + tmp_y * _max_z_id + tmp_z;
                            is_inside *= inside_data[idx_tmp];
                        }
                        else
                            is_inside = 0;   
                    }
                }
            }

            if( is_inside < 1 ) // this grid is actually outside the cube
            {
                cube_outside_grid_x.push_back(cube_grid_x[i]);     
                cube_outside_grid_y.push_back(cube_grid_y[i]);     
                cube_outside_grid_z.push_back(cube_grid_z[i]);     
            }
        }
    }
    
    for(int i = 0; i < (int)cube_outside_grid_x.size(); i++)
    {
        inside_data[cube_outside_grid_x[i] * _max_yz_id + cube_outside_grid_y[i] * _max_z_id + cube_outside_grid_z[i]] = (uint8_t)0;
    }

    insideFlagUpload();      

// ####
    // GPU-based convex clustering
    int init_cluster_grid_num = cube_outside_grid_x.size();
    int active_grid_num = init_cluster_grid_num;

    //ROS_WARN("active_grid_num, after cube inflation, is : %d", active_grid_num);

    for(int i = 0; i < init_cluster_grid_num; i++)
    {
        cluster_xyz_id[3 * i]     = cube_outside_grid_x[i];
        cluster_xyz_id[3 * i + 1] = cube_outside_grid_y[i];
        cluster_xyz_id[3 * i + 2] = cube_outside_grid_z[i];

        active_xyz_id[3 * i]      = cube_outside_grid_x[i];
        active_xyz_id[3 * i + 1]  = cube_outside_grid_y[i];
        active_xyz_id[3 * i + 2]  = cube_outside_grid_z[i];
    }

    // degenerate case,
    if( abs(vertex_idx[0 + 7]  - vertex_idx[0 + 1] ) == 0 || abs(vertex_idx[8 + 7]  - vertex_idx[8 + 1] ) == 0 || abs(vertex_idx[16 + 7] - vertex_idx[16 + 1]) == 0 )
    {   
        for(int i = 0; i < init_cluster_grid_num; i++)
        {
            cluster_x_idx.push_back(cluster_xyz_id[3 * i]);
            cluster_y_idx.push_back(cluster_xyz_id[3 * i + 1]);
            cluster_z_idx.push_back(cluster_xyz_id[3 * i + 2]);
        }
        return;
    }

    time_1_poly = ros::Time::now();
    double time_cuda_poly     = 0.0;
    double time_upload_poly   = 0.0;
    double time_download_poly = 0.0;

    ros::Time time_upload_init1 = ros::Time::now();
    hipMemcpy(d_cluster_xyz_id, cluster_xyz_id, 3 * init_cluster_grid_num * sizeof(int), hipMemcpyHostToDevice);   
    ros::Time time_upload_init2 = ros::Time::now();
    time_upload_poly += (time_upload_init2 - time_upload_init1).toSec();

    int cluster_grid_num = init_cluster_grid_num;

if  (_is_gpu_on_stage_2)
    polytopeCluster_gpu(cluster_grid_num, active_grid_num, time_upload_poly, time_download_poly, time_cuda_poly);
else 
    polytopeCluster_cpu( cluster_grid_num, active_grid_num );
    time_2_poly = ros::Time::now();

if ( DEBUG_INFO_VERBOSE_LEVEL_1 && _is_gpu_on_stage_1 )
{
    ROS_INFO("<Inflation> Time in cuda kernel: %f", time_cuda_cube );
    ROS_INFO("<Inflation> Time in data upload: %f", time_upload_cube );
    ROS_INFO("<Inflation> Time in data download: %f", time_download_cube );
}

if ( DEBUG_INFO_VERBOSE_LEVEL_1 && _is_gpu_on_stage_2 )
{
    ROS_INFO("<Clustering> Time in cuda kernel: %f",   time_cuda_poly     );
    ROS_INFO("<Clustering> Time in data upload: %f",   time_upload_poly   );
    ROS_INFO("<Clustering> Time in data download: %f", time_download_poly );
}

    ros::Time time_finish_gpu_computing = ros::Time::now();

    for(int i = 0; i < cluster_grid_num; i++)
    {
        cluster_x_idx.push_back(cluster_xyz_id[3 * i]);
        cluster_y_idx.push_back(cluster_xyz_id[3 * i + 1]);
        cluster_z_idx.push_back(cluster_xyz_id[3 * i + 2]);
    }

    cout<<"[polyhedron_generator]{GPU} finish gpu cluster"<<endl;
}   