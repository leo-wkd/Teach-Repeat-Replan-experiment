#include "hip/hip_runtime.h"
#include "polyhedron_generator/cluster_engine.cuh"
#include "polyhedron_generator/cluster_engine_cpu.h"

using namespace std;

__device__ 
double signum(const int & x)
{
    return x == 0 ? 0 : x < 0 ? -1 : 1;
}

__device__ 
double mod(const double & value, const double & modulus)
{
    return fmod(fmod(value, modulus) + modulus,  modulus);
}

__device__  
double intbound(double s, int ds)
{
    // Find the smallest positive t such that s+t*ds is an integer.
    if (ds == 0)
    {
        return 99999.0; 
    }
    else if (ds < 0)
    {
        return intbound(-s, -ds);
    }
    else
    {
        s = mod(s, 1.0f);
        return (1-s)/ds;
    }
}

__global__ void paraResultCheck(
            bool * d_result, bool * d_can_can_result, bool * d_can_clu_result,
            const int candidate_grid_num, const int cluster_grid_num )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if( tid < candidate_grid_num)
    {   
        int check_grid_num = candidate_grid_num + cluster_grid_num;

        int n = tid + 1;
        int can_can_cnt_bias = n * (n - 1) / 2;

        for(int i = 0; i < tid; i++)
        {   
            d_can_can_result[can_can_cnt_bias + i] = d_result[tid * check_grid_num + i];
                //d_can_can_result[can_can_cnt_bias + i] = true;
        }

        for(int i = 0; i < cluster_grid_num; i++)
        {   
            if(d_result[tid * check_grid_num + candidate_grid_num + i] == false) 
            {   
                d_can_clu_result[tid] = false;
                return;
            }
        }

        d_can_clu_result[tid]   = true;
    }
}

#if 1
__global__ void paraConvexTest(
            const uint8_t * d_map_data,     const uint8_t * d_inside_data,     
            const int * d_candidate_xyz_id, const int * d_cluster_xyz_id,   bool * d_result,
            const int map_yz_size,          const int map_z_size,
            const int candidate_grid_num,   const int cluster_grid_num )
{
    //__syncthreads();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // for every candidate grid, we need to check it's ray to these many grids: all already clustered grids + all possible candidates
    int check_grid_num = candidate_grid_num + cluster_grid_num;

    if( tid < candidate_grid_num * check_grid_num ) 
    {   
        int candidate_id = tid / check_grid_num;                // the ID of the current candidate
        int target_id    = tid - candidate_id * check_grid_num; // the ID of the current target

        int endX, endY, endZ;
        if( target_id < candidate_grid_num )
        {
            if( target_id >= candidate_id )   
                return;
            else
            {
                endX = d_candidate_xyz_id[3 * target_id];
                endY = d_candidate_xyz_id[3 * target_id + 1];
                endZ = d_candidate_xyz_id[3 * target_id + 2];
            }
        }
        else
        {
            endX = d_cluster_xyz_id[3 * (target_id - candidate_grid_num)];
            endY = d_cluster_xyz_id[3 * (target_id - candidate_grid_num) + 1];
            endZ = d_cluster_xyz_id[3 * (target_id - candidate_grid_num) + 2];   
        }

        d_result[tid] = true;

        int x, y, z;
        int dx, dy, dz, stepX, stepY, stepZ;
        double tMaxX, tMaxY, tMaxZ;
        double tDeltaX, tDeltaY, tDeltaZ;

        x    = d_candidate_xyz_id[3 * candidate_id];
        y    = d_candidate_xyz_id[3 * candidate_id + 1];
        z    = d_candidate_xyz_id[3 * candidate_id + 2];

        dx = endX - x;
        dy = endY - y;
        dz = endZ - z;

        stepX = signum(dx);
        stepY = signum(dy);
        stepZ = signum(dz);

        tMaxX = intbound(0.5, dx);
        tMaxY = intbound(0.5, dy);
        tMaxZ = intbound(0.5, dz);

        // The change in t when taking a step (always positive).
        tDeltaX = ((double)stepX) / dx;
        tDeltaY = ((double)stepY) / dy;
        tDeltaZ = ((double)stepZ) / dz;

        while (true)
        {   
            if(x == endX && y == endY && z == endZ)
                break;
            
            if (tMaxX < tMaxY)
            {
                if (tMaxX < tMaxZ)
                {
                    // Update which cube we are now in.
                    x += stepX;
                    tMaxX += tDeltaX;
                }
                else
                {
                    z += stepZ;
                    tMaxZ += tDeltaZ;
                }
            }
            else
            {
                if (tMaxY < tMaxZ)
                {
                    y += stepY;
                    tMaxY += tDeltaY;
                }
                else
                {
                    z += stepZ;
                    tMaxZ += tDeltaZ;
                }
            }

            int idx = x * map_yz_size + y * map_z_size + z;
            
            if( d_inside_data[ idx ] > 0 )
                return;
            
            if( x == endX && y == endY && z == endZ )
                break;
            
            if( d_map_data[ idx ] > 0 ) // here we find a obstacle on the ray
            {   
                //if((x * map_yz_size + y * map_z_size + z)<=0 || (x * map_yz_size + y * map_z_size + z) >= grid_num)
                //    printf("index: %d,", x * map_yz_size + y * map_z_size + z);
                d_result[tid] = false;
            }
        }
    }
}
#endif

__global__ void paraCubeInflation( 
            int dir, int inf_step, // the direction of the current inflation
            uint8_t  * d_map_data, int map_yz_size, int map_z_size, 
            int  * d_vertex_idx, 
            bool * d_inflate_result )
{
    //__syncthreads();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_x, grid_y, grid_z, grid_num;
    *d_inflate_result = true;

    switch(dir) 
    {   
        case 0: //direction Y -
            grid_x = (d_vertex_idx[0  + 0] - d_vertex_idx[0  + 3] + 1);
            grid_z = (d_vertex_idx[16 + 0] - d_vertex_idx[16 + 4] + 1);
            grid_num = grid_x * grid_z;

            if(tid < grid_num)
            {   
                int id_x = tid / grid_z;
                int id_z = tid - id_x * grid_z; 

                id_x += d_vertex_idx[0  + 3];
                id_z += d_vertex_idx[16 + 4];

                for(int i = 1; i <= inf_step; i++)
                {
                    int id_y = d_vertex_idx[8 + 0] - i;

                    if( d_map_data   [ id_x * map_yz_size + id_y * map_z_size + id_z ] > 0 )
                    {   
                        //printf("DIR: %d, %d, %d, %d \n", dir, id_x, id_y, id_z);
                        *d_inflate_result = false;
                    }
                }
            }
            break;
        case 1: //direction Y +
            grid_x = (d_vertex_idx[0  + 1] - d_vertex_idx[0  + 2] + 1);
            grid_z = (d_vertex_idx[16 + 1] - d_vertex_idx[16 + 5] + 1);
            grid_num = grid_x * grid_z;

            if(tid < grid_num)
            {
                int id_x = tid / grid_z;
                int id_z = tid - id_x * grid_z; 

                id_x += d_vertex_idx[0  + 2];
                id_z += d_vertex_idx[16 + 5];
                
                for(int i = 1; i <= inf_step; i++)
                {
                    int id_y = d_vertex_idx[8 + 1] + i;

                    if( d_map_data   [ id_x * map_yz_size + id_y * map_z_size + id_z ] > 0 )
                    {   
                        //printf("DIR: %d, %d, %d, %d \n", dir, id_x, id_y, id_z);
                        *d_inflate_result = false;
                    }
                }
            }
            break; 
        case 2: //direction X -
            grid_y = (d_vertex_idx[8  + 2] - d_vertex_idx[8  + 3] + 1);
            grid_z = (d_vertex_idx[16 + 3] - d_vertex_idx[16 + 7] + 1);
            grid_num = grid_y * grid_z;

            if(tid < grid_num)
            {
                int id_y = tid / grid_z;
                int id_z = tid - id_y * grid_z; 

                id_y += d_vertex_idx[8 + 3];
                id_z += d_vertex_idx[16 + 7];

                for(int i = 1; i <= inf_step; i++)
                {
                    int id_x = d_vertex_idx[0 + 3] - i;

                    if( d_map_data   [ id_x * map_yz_size + id_y * map_z_size + id_z ] > 0 )
                    {   
                        //printf("DIR: %d, %d, %d, %d \n", dir, id_x, id_y, id_z);
                        *d_inflate_result = false;
                    }
                }
            }
            break; 
        case 3: //direction X +
            grid_y = (d_vertex_idx[8  + 1] - d_vertex_idx[8  + 0] + 1);
            grid_z = (d_vertex_idx[16 + 0] - d_vertex_idx[16 + 4] + 1);
            grid_num = grid_y * grid_z;

            if(tid < grid_num)
            {
                int id_y = tid / grid_z;
                int id_z = tid - id_y * grid_z; 
                
                id_y += d_vertex_idx[8  + 0];
                id_z += d_vertex_idx[16 + 4];

                for(int i = 1; i <= inf_step; i++)
                {
                    int id_x = d_vertex_idx[0 + 0] + i;
                    if( d_map_data   [ id_x * map_yz_size + id_y * map_z_size + id_z ] > 0 )
                    {   
                        //printf("DIR: %d, %d, %d, %d \n", dir, id_x, id_y, id_z);
                        *d_inflate_result = false;
                    }
                }
            }
            break; 
        case 4: //direction Z -
            grid_y = (d_vertex_idx[8 + 5] - d_vertex_idx[8 + 4] + 1);
            grid_x = (d_vertex_idx[0 + 4] - d_vertex_idx[0 + 7] + 1);
            grid_num = grid_y * grid_x;

            if(tid < grid_num)
            {   
                int id_y = tid / grid_x;
                int id_x = tid - id_y * grid_x;
                
                id_y += d_vertex_idx[8 + 4];
                id_x += d_vertex_idx[0 + 7];
                
                for(int i = 1; i <= inf_step; i++)
                {
                    int id_z = d_vertex_idx[16 + 4] - i;

                    if( d_map_data   [ id_x * map_yz_size + id_y * map_z_size + id_z ] > 0 )
                    {   
                        //printf("DIR: %d, %d, %d, %d \n", dir, id_x, id_y, id_z);
                        *d_inflate_result = false;
                    }
                }
            }
            break; 
        case 5: //direction Z +  
            grid_y = (d_vertex_idx[8 + 1] - d_vertex_idx[8 + 0] + 1);
            grid_x = (d_vertex_idx[0 + 0] - d_vertex_idx[0 + 3] + 1);
            grid_num = grid_y * grid_x;

            if(tid < grid_num)
            {   
                int id_y = tid / grid_x;
                int id_x = tid - id_y * grid_x; 
                
                id_y += d_vertex_idx[8 + 0];
                id_x += d_vertex_idx[0 + 3];
                
                for(int i = 1; i <= inf_step; i++)
                {
                    int id_z = d_vertex_idx[16 + 0] + i;

                    if( d_map_data   [ id_x * map_yz_size + id_y * map_z_size + id_z ] > 0 )
                    {   
                        //printf("DIR: %d, %d, %d, %d \n", dir, id_x, id_y, id_z);
                        *d_inflate_result = false;
                    }
                }
            }
            break;  
        default: 
            break; 
    }
}